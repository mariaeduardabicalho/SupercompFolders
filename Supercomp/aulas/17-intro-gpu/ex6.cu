#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <iostream>

int main() {

    thrust::device_vector<double> AAPL;
    thrust::device_vector<double> MSFT;
    thrust::device_vector<double> MEAN_DIF(2518,0);
    double stocks_AAPL, stocks_MSFT, mean;

    for(int i =0; i < 2518; i++){
        std::cin >> stocks_AAPL >> stocks_MSFT;
        AAPL.push_back(stocks_AAPL);
        MSFT.push_back(stocks_MSFT);
    }


    thrust::transform(AAPL.begin(), AAPL.end(), MSFT.begin(), MEAN_DIF.begin(), thrust::minus<double>());
    double val = thrust::reduce(MEAN_DIF.begin(), MEAN_DIF.end());
    mean = val/2517;
    std::cout << "Média: " << fabs(mean) << "\n";

}