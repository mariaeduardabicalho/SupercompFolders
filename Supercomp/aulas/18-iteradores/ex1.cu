#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <iostream>

int main() {

    thrust::device_vector<double> AAPL;
    thrust::device_vector<double> MSFT;
    thrust::device_vector<double> MEAN_DIF(2518,0);
    double stocks_AAPL, stocks_MSFT, mean;

    for(int i =0; i < 2518; i++){
        std::cin >> stocks_AAPL >> stocks_MSFT;
        AAPL.push_back(stocks_AAPL);
        MSFT.push_back(stocks_MSFT);
    }


    thrust::transform(AAPL.begin(), AAPL.end(), MSFT.begin(), MEAN_DIF.begin(), thrust::minus<double>());
    double val = thrust::reduce(MEAN_DIF.begin(), MEAN_DIF.end());
    mean = val/2517;
    std::cout << "Média: " << fabs(mean) << "\n";

    thrust::device_vector<double> means(2518,mean);
    thrust::device_vector<double> dps(2518,0);
    thrust::device_vector<double> vars(2518,0);

    thrust::transform(means.begin(), means.end(), MEAN_DIF.begin(), dps.begin(), thrust::minus<double>());
    thrust::transform(dps.begin(), dps.end(), vars.begin(),thrust::square<double>());
    double v = thrust::reduce(vars.begin(), vars.end());
    double var = v/ 2517;

    std::cout << "variancia : " << fabs(var) << "\n";

}