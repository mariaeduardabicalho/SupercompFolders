#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <iostream>
#include <math.h> 
#include <thrust/iterator/constant_iterator.h>

struct variancia
{
    double media;
    int N;
    variancia(double m, int n): media(m), N(n){};
    __host__ __device__

    double operator()(const double& x) {
        return pow((x - media),2)/N;
        
    }
};

int main() {
    thrust::device_vector<double> AAPL;
    
    double stocks_AAPL, mean, var;

    while(std::cin){
        std::cin >> stocks_AAPL;
        AAPL.push_back(stocks_AAPL);
    }

    int N = AAPL.size();

    double val = thrust::reduce(AAPL.begin(), AAPL.end());
    mean = val/2517;
    std::cout << "Média: " << mean << "\n";


    var = thrust::transform_reduce(AAPL.begin(), AAPL.end(), variancia(mean, N),0.0, thrust::plus<double>());

    std::cout << "Variância: " << var << "\n";

}