#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <iostream>
#include <math.h> 
#include <thrust/iterator/constant_iterator.h>

struct media {
    int cols;
    unsigned char *pixels;

    media(inr cols, unsigned char *pixels) : cols(cols), pixels(pixels) {};

    __host__ __device__
    unsigned char operator() (const int &i){
        int x = i % cols;
        int y = i / cols;
        return (pixels[i]+ pixels[i+1] +pixels[i-1])/3;
        }
};

int main(int argc, char *argv[]){
    Imagem im = Imagem::read(std::string(argv[1]));
    
    //COPIAR P/ GPU
    thrust::device_vector<unsigned char> pixels_gpu (im.pixels,im.pixels + im.total_size);
    thrust::device_vector<unsigned char> pixels_out (im.total_size);

    //transform
    media med(im.cols,pixels_gpu.data().get());
    thrust::transform(thrust::make_counting_iterator<int>(0),
                        thrust::make_counting_iterator<int>(im.total_size),
                        pixels_out.begin(),med);

    //copia de volta p cpu
    thrust::copy(pixels_out.begin(),pixels_out.end(), im.pixels);

    im.write("out.pgm");

    return 0;
}



}